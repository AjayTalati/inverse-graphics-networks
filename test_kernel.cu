#include "hip/hip_runtime.h"

#include <stdio.h>
extern "C" {
#include "test_kernel.h"
}

__global__ void cuda_dot(double a, double *help)
{
   *help=2*a;
}

//kernel calling function
extern "C" 
void cuda_GMRESfunc(double a)
{
	double b;

	double *dev_a;
	double *res;

	hipMemcpy(dev_a, &a, sizeof(double), hipMemcpyHostToDevice );
	cuda_dot<<< 1, 1 >>>(*dev_a, res );
	hipMemcpy(&b, res, sizeof(double), hipMemcpyDeviceToHost );
}    


